#include <stdio.h>
#include <iostream>
#include <algorithm>
#include <fstream>
#include <random>
#include <chrono>
#include "include/preprocessor.hpp"
#include "include/csv.hpp"
#include "include/cudadebug.cuh"
#include "include/encoding.cuh"
#include "include/kernels.cuh"
#include <hipblas.h>
// #include <hip/hip_runtime.h>

std::string x_train_path = "../dataset/isolet_train_trainX.bin";
std::string y_train_path = "../dataset/isolet_train_trainY.bin";

void datasetBinaryRead(std::vector<int> &data, std::string path){
	std::ifstream file_(path, std::ios::in | std::ios::binary);
	// assert(file_.is_open() && "Couldn't open file!");
	int32_t size;
	file_.read((char*)&size, sizeof(size));
	int32_t temp;
	for(int i = 0; i < size; i++){
		file_.read((char*)&temp, sizeof(temp));
		data.push_back(temp);
	}
	file_.close();
}

// #define USE_DOT_SIMILARITY
#define USE_COS_SIMILARITY

#define USE_DOT_ENCODING
//#define USE_LVID_ENCODING

int main(int argc, char* argv[]) {
    //fclose(fopen("cluster_results.csv", "w"));  //clear file first
    //fclose(fopen("train_labels.csv", "w"));  //clear file first
    // ./main [TRAIN dataset path] [DIM] [ITER] [Q]
    // Example:
    // ./main datasets/UCIHAR/UCIHAR_train.choir_dat 10000 20 100

    int nFeatures_train;
    int nClasses_train;  // nFeatures is same as x_train[0].size()
    //std::vector<std::vector<float>> x_train;
    std::vector<int> x_train_int;
    std::vector<int> y_train;



    datasetBinaryRead(x_train_int, x_train_path);
    datasetBinaryRead(y_train, y_train_path);
    
    nFeatures_train = 617;
    nClasses_train = 26;
    // int train_set_num = 6238;

    std::vector<float> x_train_flat = std::vector<float>(x_train_int.begin(), x_train_int.end());
    // readChoirDat(argv[1], nFeatures_train, nClasses_train, x_train, y_train);
    //readFCPSTrainDat(argv[1], nFeatures_train, nClasses_train, x_train, y_train);


    // x_train_flat
    //l2norm(x_train);
    //std::vector<float> x_train_flat = flatten(x_train);

    // base_creation: linear
    int dim = 2048;
    int iter_num = atoi(argv[1]);


    std::cout << "epochs:" << iter_num << std::endl;

    int Q = 32; // Hopefully unused. atoi(argv[42]); // nLevel

    // int K = atoi(argv[5]);
    int K = nClasses_train;

    int train_set_num = (x_train_flat.size() - nClasses_train) / 617; 

    std::cout << "train set num: " << train_set_num << std::endl;
    int base_size = nFeatures_train * dim;
    size_t train_encode_size = train_set_num * dim;

    std::cout << "train encode isze? : " << train_encode_size << std::endl;
    std::cout << train_set_num << " " << nFeatures_train << std::endl;

    // K equals to the number of classes

    // generate bases
    std::vector<float> bases;  // flattened
    std::vector<float> base_v1(dim/2, 1);
    std::vector<float> base_v2(dim/2, -1);
    base_v1.insert(base_v1.end(), base_v2.begin(), base_v2.end());
    // obtain a time-based seed
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    for(int i = 0 ; i < nFeatures_train; i++) {
        std::shuffle(base_v1.begin(), base_v1.end(), std::default_random_engine(seed));
        bases.insert(bases.end(), base_v1.begin(), base_v1.end());
    }

    // generate level
    std::vector<float> level_base(base_v1);
    std::vector<float> level_hvs;
    for (int q = 0; q <= Q; ++q) {
        int flip = (int) (q/float(Q) * dim) / 2;
        std::vector<float> level_hv(level_base);
        // + flip will transform (flip) number of elements
        std::transform(level_hv.begin(), level_hv.begin() + flip, level_hv.begin(), bind2nd(std::multiplies<float>(), -1)); 
        level_hvs.insert(level_hvs.end(), level_hv.begin(), level_hv.end());
    }

    // generate id
    std::shuffle(level_base.begin(), level_base.end(), std::default_random_engine(seed));  // use this as id_base
    std::vector<float> id_hvs(level_base);  // f=0
    for (int f = 1; f < nFeatures_train; ++f) {
        std::rotate(level_base.begin(), level_base.begin() + 1, level_base.end());
        id_hvs.insert(id_hvs.end(), level_base.begin(), level_base.end());
    }

    // rand gen K number of 0 ~ train_set_num - 1
    std::vector<int> rand_weights(train_set_num);
    std::iota(std::begin(rand_weights), std::end(rand_weights), 0);                                                                                        
    std::shuffle(rand_weights.begin(), rand_weights.end(), std::default_random_engine(seed));  // random order
    rand_weights.resize(K);

    int* cluster_results;
    cluster_results = (int*)malloc(train_set_num * sizeof(int));

    //////////////////////////////////////////////////////////////////////////
    // GPU LOAD
    float* d_bases = NULL;
    float* d_x_train = NULL;
    float* d_hvs_train = NULL;

#ifdef USE_COS_SIMILARITY
    float* d_train_norm = NULL;
    float* d_weights_norm = NULL;
#endif
    HANDLE_ERROR(hipMalloc((void **)&d_bases, base_size * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void **)&d_x_train, x_train_flat.size() * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void **)&d_hvs_train, 4 * train_encode_size * sizeof(float)));
    
#ifdef USE_COS_SIMILARITY
    HANDLE_ERROR(hipMalloc((void **)&d_train_norm, train_set_num * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void **)&d_weights_norm, K * sizeof(float)));
#endif

    HANDLE_ERROR(hipMemcpy(d_bases, bases.data(), base_size * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_x_train, x_train_flat.data(), x_train_flat.size() * sizeof(float), hipMemcpyHostToDevice));

    //id level
    float* d_level_hvs = NULL;
    float* d_id_hvs = NULL;
    HANDLE_ERROR(hipMalloc((void **)&d_level_hvs, level_hvs.size() * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void **)&d_id_hvs, id_hvs.size() * sizeof(float)));
    HANDLE_ERROR(hipMemcpy(d_level_hvs, level_hvs.data(), level_hvs.size() * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_id_hvs, id_hvs.data(), id_hvs.size() * sizeof(float), hipMemcpyHostToDevice));


    std::cout << "class: " << nClasses_train << std::endl;
    float* d_weights = NULL;
    HANDLE_ERROR(hipMalloc((void **)&d_weights, K * dim * sizeof(float)));

    float* d_guess_table = NULL;
    int* d_y_pred = NULL;
    HANDLE_ERROR(hipMalloc((void **)&d_guess_table, K * train_set_num * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void **)&d_y_pred, train_set_num * sizeof(int)));

    hipEvent_t start, stop1, stop2;
    hipError_t err = hipSuccess;

    hipEventCreate(&start);
    hipEventCreate(&stop1);
    hipEventCreate(&stop2);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    const float alpha = 1;
    const float beta = 0;

    hipEventRecord(start);
    printf("Starting Encoding Stage...\n");

#ifdef USE_DOT_ENCODING
    // Encode stage: Linear
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
               dim, train_set_num, nFeatures_train, 
               &alpha, d_bases, dim, 
               d_x_train, nFeatures_train, &beta, 
               d_hvs_train, dim);
    hipDeviceSynchronize();
#endif

#ifdef USE_LVID_ENCODING
    dim3 encodeblocksTrain((dim + N_THREADS - 1) / N_THREADS, train_set_num);
    dim3 encodeTPB(N_THREADS, 1, 1);

    int level_stride = dim * 4;
    int id_stride = dim * 4;
    int fm_stride = nFeatures_train * 4;

    encodeLevelId<<<encodeblocksTrain, encodeTPB>>>(d_level_hvs, d_id_hvs, d_x_train, d_hvs_train, level_stride, 
                                                id_stride, fm_stride, train_set_num, nFeatures_train, Q, dim);
#endif

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch kernel (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Initialize class HVs
    for (int ii = 0; ii < K; ++ii) {
        HANDLE_ERROR(hipMemcpy(d_weights + ii * dim, d_hvs_train + rand_weights[ii] * dim, dim * sizeof(float), hipMemcpyDeviceToDevice));
    }

    hipEventRecord(stop1);
    hipEventSynchronize(stop1);

    // Clustering stage
    // make_guess and create guess table
    printf("Clustering stage...\n");

#ifdef USE_COS_SIMILARITY
    normMatRow<<<(train_set_num + N_THREADS - 1) / N_THREADS, N_THREADS>>>(d_train_norm, d_hvs_train, train_set_num, dim);
#endif

    for (int iter = 0; iter < iter_num; ++iter) {  // Retraining, Different from max_iter
        hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                    K, train_set_num, dim, 
                    &alpha, d_weights, dim, 
                    d_hvs_train, dim, &beta, 
                    d_guess_table, K);   // np.dot
        hipDeviceSynchronize();

#ifdef USE_COS_SIMILARITY
        normMatRow<<<(K + N_THREADS - 1) / N_THREADS, N_THREADS>>>(d_weights_norm, d_weights, K, dim);

        guessVecGenCompareCosine<<<(train_set_num + N_THREADS - 1)/N_THREADS, N_THREADS>>>(d_y_pred, d_weights_norm, d_train_norm, d_guess_table, train_set_num, K);
#endif

#ifdef USE_DOT_SIMILARITY
        guessVecGenCompareDot<<<(train_set_num + N_THREADS - 1)/N_THREADS, N_THREADS>>>(d_y_pred, d_guess_table, train_set_num, K);
#endif

        updateClassHV<<<(dim + N_THREADS - 1) / N_THREADS, N_THREADS>>>(d_hvs_train, d_weights, d_y_pred, train_set_num, dim);
    }

    HANDLE_ERROR(hipMemcpy(cluster_results, d_y_pred, train_set_num * sizeof(int), hipMemcpyDeviceToHost));
    hipEventRecord(stop2);
    hipEventSynchronize(stop2);

    hipblasDestroy(handle);
    HANDLE_ERROR(hipFree(d_bases));
    HANDLE_ERROR(hipFree(d_x_train));
    HANDLE_ERROR(hipFree(d_hvs_train));
    HANDLE_ERROR(hipFree(d_weights));

    HANDLE_ERROR(hipFree(d_guess_table));
    HANDLE_ERROR(hipFree(d_y_pred));

#ifdef USE_COS_SIMILARITY
    HANDLE_ERROR(hipFree(d_train_norm));
    HANDLE_ERROR(hipFree(d_weights_norm));
#endif

    HANDLE_ERROR(hipFree(d_level_hvs));
    HANDLE_ERROR(hipFree(d_id_hvs));

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop1);
    printf("GPU Execution time (Encoding): %f\n", milliseconds);
    hipEventElapsedTime(&milliseconds, stop1, stop2);
    printf("GPU Execution time (Clustering): %f\n", milliseconds);

    // write file
    std::ofstream out("cluster_results.csv");
    for (int idx = 0; idx < train_set_num; ++idx) {
        out << cluster_results[idx] <<',';
    }
    out << '\n';
    
    std::ofstream out_gnd("train_labels.csv");
    for (auto& data : y_train) {
        out_gnd << data <<',';
    }
    out_gnd << '\n';
    
    hipEventDestroy(start);
    hipEventDestroy(stop1);
    hipEventDestroy(stop2);

    free(cluster_results);

    return 0;
}
